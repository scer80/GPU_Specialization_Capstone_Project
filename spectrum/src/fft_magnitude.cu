#include "hip/hip_runtime.h"
#include <iostream>
#include <hipfft/hipfft.h>
#include "fft_magnitude.h"


const char *cufftGetErrorString(hipfftResult error)
{
    switch (error)
    {
        case HIPFFT_SUCCESS:
            return "HIPFFT_SUCCESS";
        case HIPFFT_INVALID_PLAN:
            return "HIPFFT_INVALID_PLAN";
        case HIPFFT_INVALID_VALUE:
            return "HIPFFT_INVALID_VALUE";
        case HIPFFT_INTERNAL_ERROR:
            return "HIPFFT_INTERNAL_ERROR";
        case HIPFFT_EXEC_FAILED:
            return "HIPFFT_EXEC_FAILED";
        case HIPFFT_SETUP_FAILED:
            return "HIPFFT_SETUP_FAILED";
        case HIPFFT_NOT_SUPPORTED:
            return "HIPFFT_NOT_SUPPORTED";
        case CUFFT_LICENSE_ERROR:
            return "CUFFT_LICENSE_ERROR";
        default:
            return "Unknown CUFFT error";
    }
}


__global__ void calculate_magnitude(hipfftComplex *d_input, float *d_output, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float real = d_input[idx].x;
        float imag = d_input[idx].y;
        d_output[idx] = sqrtf(real * real + imag * imag);
    }
}


hipError_t compute_fft_magnitude(float *h_input, float *h_output, int n) {
    hipfftHandle plan;
    hipError_t cudaStatus;
    hipfftResult cufftStatus;

    // Allocate device memory
    float *d_input = nullptr;
    hipfftComplex *d_output = nullptr;
    cudaStatus = hipMalloc((void **) &d_input, n * sizeof(float));
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMalloc failed for input: " << hipGetErrorString(cudaStatus) << std::endl;
        return cudaStatus;
    }
    cudaStatus = hipMalloc((void **) &d_output, (n/2 + 1) * sizeof(hipfftComplex));
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMalloc failed for output: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_input);
        return cudaStatus;
    }

    // Create cuFFT plan
    cufftStatus = hipfftPlan1d(&plan, n, HIPFFT_R2C, 1);
    if (cufftStatus != HIPFFT_SUCCESS) {
        std::cerr << "hipfftPlan1d failed: " << cufftGetErrorString(cufftStatus) << std::endl;
        hipFree(d_input);
        hipFree(d_output);
        return hipErrorUnknown;
    }

    // Copy input data to device
    cudaStatus = hipMemcpy(d_input, h_input, n * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMemcpy failed to copy input to device: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_input);
        hipFree(d_output);
        hipfftDestroy(plan);
        return cudaStatus;
    }

    // Execute FFT
    cufftStatus = hipfftExecR2C(plan, d_input, d_output);
    if (cufftStatus != HIPFFT_SUCCESS) {
        std::cerr << "hipfftExecR2C failed: " << cufftGetErrorString(cufftStatus) << std::endl;
        hipFree(d_input);
        hipFree(d_output);
        hipfftDestroy(plan);
        return hipErrorUnknown;
    }

    // Calculate magnitude and copy to host
    float *d_magnitude = nullptr;
    cudaStatus = hipMalloc((void **) &d_magnitude, (n/2 + 1) * sizeof(float));
     if (cudaStatus != hipSuccess) {
        std::cerr << "hipMalloc failed for magnitude: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_input);
        hipFree(d_output);
        hipfftDestroy(plan);
        return cudaStatus;
    }

    dim3 dimBlock(256);
    dim3 dimGrid((n/2 + 1 + dimBlock.x - 1) / dimBlock.x);

    calculate_magnitude<<<dimGrid, dimBlock>>>(d_output, d_magnitude, n/2 + 1);
    hipError_t kernelErr = hipGetLastError();
    if (kernelErr != hipSuccess) {
        std::cerr << "calculate_magnitude launch failed: " << hipGetErrorString(kernelErr) << std::endl;
        hipFree(d_input);
        hipFree(d_output);
        hipFree(d_magnitude);
        hipfftDestroy(plan);
        return hipErrorUnknown;
    }

    cudaStatus = hipMemcpy(h_output, d_magnitude, (n/2 + 1) * sizeof(float), hipMemcpyDeviceToHost);    
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMemcpy failed to copy output to host: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_input);
        hipFree(d_output);
        hipFree(d_magnitude);
        hipfftDestroy(plan);
        return cudaStatus;
    }

    // Clean up
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_magnitude);
    hipfftDestroy(plan);

    return hipSuccess;
}
